#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <unordered_set>
#include <algorithm>
#include <cstring>

#define MAX_LINES 1024
#define MAX_LINE_LENGTH 256

// 转小写工具（设备端）
__device__ void to_lower(char* str) {
    for (int i = 0; i < MAX_LINE_LENGTH; i++) {
        char c = str[i];
        if (c == '\0') break;
        if (c >= 'A' && c <= 'Z') {
            str[i] = c + 32;
        }
    }
}

// 字符串比较（设备端）
__device__ bool str_equal(const char* a, const char* b) {
    for (int i = 0; i < MAX_LINE_LENGTH; ++i) {
        if (a[i] != b[i]) return false;
        if (a[i] == '\0') return true;
    }
    return true;
}

// 核函数：并行判断是否重复
__global__ void mark_unique(char lines[][MAX_LINE_LENGTH], int line_count, bool* keep_flags) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= line_count) return;

    // 跳过空行
    if (lines[idx][0] == '\0') {
        keep_flags[idx] = false;
        return;
    }

    // 转为小写（就地）
    to_lower(lines[idx]);

    // 和前面所有行比较
    for (int i = 0; i < idx; ++i) {
        if (lines[i][0] == '\0') continue;
        if (str_equal(lines[i], lines[idx])) {
            keep_flags[idx] = false;
            return;
        }
    }

    keep_flags[idx] = true;
}

int main() {
    // 读取文件
    std::ifstream infile("input.txt");
    std::vector<std::string> host_lines;
    std::string line;

    while (std::getline(infile, line) && host_lines.size() < MAX_LINES) {
        line.erase(0, line.find_first_not_of(" \t\r\n"));
        line.erase(line.find_last_not_of(" \t\r\n") + 1);
        if (!line.empty()) {
            host_lines.push_back(line.substr(0, MAX_LINE_LENGTH - 1));
        }
    }
    infile.close();
    int line_count = host_lines.size();

    // 拷贝数据到平面内存
    char h_lines[MAX_LINES][MAX_LINE_LENGTH] = {0};
    for (int i = 0; i < line_count; ++i) {
        std::strncpy(h_lines[i], host_lines[i].c_str(), MAX_LINE_LENGTH - 1);
    }

    // 分配 GPU 内存
    char (*d_lines)[MAX_LINE_LENGTH];
    bool* d_keep;
    hipMalloc(&d_lines, sizeof(h_lines));
    hipMalloc(&d_keep, sizeof(bool) * line_count);

    // 拷贝到设备
    hipMemcpy(d_lines, h_lines, sizeof(h_lines), hipMemcpyHostToDevice);

    // 启动核函数
    int threadsPerBlock = 256;
    int blocks = (line_count + threadsPerBlock - 1) / threadsPerBlock;
    mark_unique<<<blocks, threadsPerBlock>>>(d_lines, line_count, d_keep);

    // 拷贝结果回主机
    bool h_keep[MAX_LINES];
    hipMemcpy(h_keep, d_keep, sizeof(bool) * line_count, hipMemcpyDeviceToHost);

    // 输出
    std::cout << "去重并忽略大小写后的结果：" << std::endl;
    for (int i = 0; i < line_count; ++i) {
        if (h_keep[i]) {
            std::cout << host_lines[i] << std::endl;
        }
    }

    // 清理
    hipFree(d_lines);
    hipFree(d_keep);

    return 0;
}

